
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

__global__ void printNumber(int number)
{
  printf("%d\n", number);
}

int main()
{
  for (int i = 0; i < 5; ++i)
  {
    // printNumber<<<1, 1>>>(i);
    hipStream_t stream;       // CUDA streams are of type `cudaStream_t`.
    hipStreamCreate(&stream); // Note that a pointer must be passed to `cudaCreateStream`.
    printNumber<<<1, 1, 0, stream>>>(i);
    hipStreamDestroy(stream); // Note that a value, not a pointer, is passed to `cudaDestroyStream`.
  }
  hipDeviceSynchronize();
}

