
#include <hip/hip_runtime.h>
__global__
void deviceKernel(int *a, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] = 1;
  }
}

void hostFunction(int *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = 1;
  }
}

int main()
{

  int N = 2<<24;
  size_t size = N * sizeof(int);
  int *a;
  hipMallocManaged(&a, size);

  /*
   * Conduct experiments to learn more about the behavior of
   * `cudaMallocManaged`.
   *
   * What happens when unified memory is accessed only by the GPU?
   * What happens when unified memory is accessed only by the CPU?
   * What happens when unified memory is accessed first by the GPU then the CPU?
   * What happens when unified memory is accessed first by the CPU then the GPU?
   *
   * Hypothesize about UM behavior, page faulting specificially, before each
   * experiment, and then verify by running `nsys`.
   */
  // What happens when unified memory is accessed only by the GPU?
  deviceKernel<<<256, 256>>>(a, N);
  hipDeviceSynchronize(); // Wait for the GPU to finish before proceeding

  // What happens when unified memory is accessed only by the CPU?
  // hostFunction(a, N);

  // What happens when unified memory is accessed first by the GPU then the CPU?
  // deviceKernel<<<256, 256>>>(a, N);
  // hostFunction(a, N);

  // What happens when unified memory is accessed first by the CPU then the GPU?
  // hostFunction(a, N);
  // deviceKernel<<<256, 256>>>(a, N);

  // cudaDeviceSynchronize(); // Wait for the GPU to finish before proceeding

  hipFree(a);
}
