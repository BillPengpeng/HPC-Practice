
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

/*void loop(int N)
{
  for (int i = 0; i < N; ++i)
  {
    printf("This is iteration number %d\n", i);
  }
}*/

__global__ void loop()
{
  printf("This is iteration number %d\n", threadIdx.x);
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, only use 1 block of threads.
   */

  int N = 10;
  // loop(N);
  loop<<<1, N>>>();
  hipDeviceSynchronize();
}
