#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])

// -------------------------------------- FP32 -------------------------------------- 
// Warp Reduce Sum
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

// Block reduce sum/max/min device helper for Layer/RMS Norm/Softmax etc.
// grid 1D block 1D, grid(N/256), block(256)
template<const int NUM_THREADS=256>
__device__ __forceinline__ float block_reduce_sum_f32(float val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];
  
  val = warp_reduce_sum_f32<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val;
  __syncthreads();
  val = (lane < NUM_WARPS) ? shared[lane] : 0.0f;
  val = warp_reduce_sum_f32<NUM_WARPS>(val);
  return val;
}

// RMS Norm: x: NxK(K=256<1024), y': NxK, y'=x/rms(x) each row
// 1/rms(x) = rsqrtf( sum(x^2)/K ) each row
// grid(N*K/K), block(K<1024) N=batch_size*seq_len, K=hidden_size
// y=y'*g (g: scale)
template<const int NUM_THREADS=256>
__global__ void rms_norm_f32_kernel(float* x, float* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = bid * blockDim.x + threadIdx.x;
  const float epsilon = 1e-5f;

  __shared__ float s_variance; // shared within block
  float value = (idx < N * K) ? x[idx] : 0.0f; // load once only
  float variance = value * value;
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / (float) K + epsilon);
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  if (idx < N * K) y[idx] = (value * s_variance) * g;
}

// RMS Norm Vec4: x: NxK(K=256<1024), y': NxK, y'=x/rms(x) each row
// 1/rms(x) = rsqrtf( sum(x^2)/K ) each row
// grid(N*K/K), block(K/4<1024) N=batch_size*seq_len, K=hidden_size
// y=y'*g (g: scale)
template<const int NUM_THREADS=256/4>
__global__ void rms_norm_f32x4_kernel(float* x, float* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 4;
  const float epsilon = 1e-5f;

  __shared__ float s_variance; // shared within block
  float4 reg_x = FLOAT4(x[idx]);
  float variance = (idx < N * K) ? (reg_x.x * reg_x.x + reg_x.y * reg_x.y 
                                  + reg_x.z * reg_x.z + reg_x.w * reg_x.w) : 0.0f;
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / (float) K + epsilon);
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  float4 reg_y;
  reg_y.x = reg_x.x * s_variance * g;
  reg_y.y = reg_x.y * s_variance * g;
  reg_y.z = reg_x.z * s_variance * g;
  reg_y.w = reg_x.w * s_variance * g;
  if (idx < N * K) FLOAT4(y[idx]) = reg_y;
}

// -------------------------------------- FP16 -------------------------------------- 
// Warp Reduce Sum: Half
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ half warp_reduce_sum_f16_f16(half val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    // val = __hadd(val, __shfl_xor_sync(0xffffffff, val, mask));
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f16_f32(half val) {
  float val_f32 = __half2float(val);
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val_f32 += __shfl_xor_sync(0xffffffff, val_f32, mask);
  }
  return val_f32;
}

template<const int NUM_THREADS=256>
__device__ half block_reduce_sum_f16_f16(half val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ half shared[NUM_WARPS];
  // reduce using half dtype within warps
  val = warp_reduce_sum_f16_f16<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val;
  __syncthreads();
  val = (lane < NUM_WARPS) ? shared[lane] : __float2half(0.0f);
  val = warp_reduce_sum_f16_f16<NUM_WARPS>(val);
  return val; // half
}

template<const int NUM_THREADS=256>
__device__ float block_reduce_sum_f16_f32(half val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];
  // reduce using float dtype within warps
  float val_f32 = warp_reduce_sum_f16_f32<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val_f32;
  __syncthreads();
  val_f32 = (lane < NUM_WARPS) ? shared[lane] : 0.0f;
  val_f32 = warp_reduce_sum_f32<NUM_WARPS>(val_f32);
  return val_f32; // float
}

template<const int NUM_THREADS=256>
__global__ void rms_norm_f16_f16_kernel(half* x, half* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = bid * blockDim.x + threadIdx.x;
  const half epsilon = __float2half(1e-5f);
  const half g_      = __float2half(g);
  const half K_      = __int2half_rn(K);

  __shared__ half s_variance; // shared within block
  half value = (idx < N * K) ? x[idx] : __float2half(0.0f); // load once only
  half variance = value * value;
  variance = block_reduce_sum_f16_f16<NUM_THREADS>(variance);
  if (tid == 0) s_variance = hrsqrt(variance / (K_ + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  if (idx < N * K) y[idx] = (value * s_variance) * g_;
}

template<const int NUM_THREADS=256>
__global__ void rms_norm_f16x2_f16_kernel(half* x, half* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 2;
  const half epsilon = __float2half(1e-5f);
  const half g_      = __float2half(g);
  const half K_      = __int2half_rn(K);

  __shared__ half s_variance; // shared within block
  half2 reg_x = HALF2(x[idx]);
  half variance = (idx < N * K) ? (reg_x.x * reg_x.x 
                                 + reg_x.y * reg_x.y): __float2half(0.0f);
  variance = block_reduce_sum_f16_f16<NUM_THREADS>(variance);
  if (tid == 0) s_variance = hrsqrt(variance / (K_ + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  half2 reg_y;
  reg_y.x = reg_x.x * s_variance * g_;
  reg_y.y = reg_x.y * s_variance * g_;
  if (idx < N * K) HALF2(y[idx]) = reg_y;
}

#define HALF2_VARIANCE(reg, i) \
  (((idx + (i)) < N * K) ? ((reg).x * (reg).x + (reg).y * (reg).y) : __float2half(0.0f))

#define FLOAT2_VARIANCE(reg, i) \
  (((idx + (i)) < N * K) ? ((reg).x * (reg).x + (reg).y * (reg).y) : 0.0f)

#define HALF2_RMS_NORM(reg_y, reg_x, g) \
  (reg_y).x = (reg_x).x * s_variance * (g); (reg_y).y = (reg_x).y * s_variance * (g); 

#define FLOAT2_RMS_NORM(reg_y, reg_x, g) \
  (reg_y).x = (reg_x).x * s_variance * (g); (reg_y).y = (reg_x).y * s_variance * (g);

template<const int NUM_THREADS=256>
__global__ void rms_norm_f16x8_f16_kernel(half* x, half* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 8;
  const half epsilon = __float2half(1e-5f);
  const half g_      = __float2half(g);
  const half K_      = __int2half_rn(K);

  __shared__ half s_variance; // shared within block
  half2 reg_x_0 = HALF2(x[idx + 0]);
  half2 reg_x_1 = HALF2(x[idx + 2]);
  half2 reg_x_2 = HALF2(x[idx + 4]);
  half2 reg_x_3 = HALF2(x[idx + 6]);
  half variance = HALF2_VARIANCE(reg_x_0, 0);
  variance     += HALF2_VARIANCE(reg_x_1, 2);
  variance     += HALF2_VARIANCE(reg_x_2, 4);
  variance     += HALF2_VARIANCE(reg_x_3, 6);
  variance = block_reduce_sum_f16_f16<NUM_THREADS>(variance);
  if (tid == 0) s_variance = hrsqrt(variance / (K_ + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  // manual unroll
  half2 reg_y_0, reg_y_1, reg_y_2, reg_y_3;
  HALF2_RMS_NORM(reg_y_0, reg_x_0, g_);
  HALF2_RMS_NORM(reg_y_1, reg_x_1, g_);
  HALF2_RMS_NORM(reg_y_2, reg_x_2, g_);
  HALF2_RMS_NORM(reg_y_3, reg_x_3, g_);
  if ((idx + 0) < N * K) { HALF2(y[idx + 0]) = reg_y_0; }
  if ((idx + 2) < N * K) { HALF2(y[idx + 2]) = reg_y_1; }
  if ((idx + 4) < N * K) { HALF2(y[idx + 4]) = reg_y_2; }
  if ((idx + 6) < N * K) { HALF2(y[idx + 6]) = reg_y_3; }
}

template<const int NUM_THREADS=256>
__global__ void rms_norm_f16x8_f32_kernel(half* x, half* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 8;
  const float epsilon = 1e-5f;

  __shared__ float s_variance; // shared within block
  // manual unroll and improve L2 cache hit rate.
  // Only   L2 cache: load 32  bytes in 1 memory issue (default)
  // Enable L1 cache: load 128 bytes in 1 memory issue (-Xptxas -dlcm=ca)
  // why try fp16x8 within 1 threads? ref: https://zhuanlan.zhihu.com/p/641639133
  // 0. first, tid_0 load 32 bytes in 1 memory issue and cache data into L2 cache.
  // 1. then, tid_1,...,tid_3 hit L2 cache and load data from L2 cache directly.
  float2 reg_x_0 = __half22float2(HALF2(x[idx + 0]));
  float2 reg_x_1 = __half22float2(HALF2(x[idx + 2]));
  float2 reg_x_2 = __half22float2(HALF2(x[idx + 4]));
  float2 reg_x_3 = __half22float2(HALF2(x[idx + 6]));

  float variance = FLOAT2_VARIANCE(reg_x_0, 0);
  variance      += FLOAT2_VARIANCE(reg_x_1, 2);
  variance      += FLOAT2_VARIANCE(reg_x_2, 4);
  variance      += FLOAT2_VARIANCE(reg_x_3, 6);

  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / ((float) K + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  // manual unroll
  float2 reg_y_0, reg_y_1, reg_y_2, reg_y_3;
  FLOAT2_RMS_NORM(reg_y_0, reg_x_0, g);
  FLOAT2_RMS_NORM(reg_y_1, reg_x_1, g);
  FLOAT2_RMS_NORM(reg_y_2, reg_x_2, g);
  FLOAT2_RMS_NORM(reg_y_3, reg_x_3, g);
  if ((idx + 0)  < N * K) { HALF2(y[idx + 0])  = __float22half2_rn(reg_y_0); }
  if ((idx + 2)  < N * K) { HALF2(y[idx + 2])  = __float22half2_rn(reg_y_1); }
  if ((idx + 4)  < N * K) { HALF2(y[idx + 4])  = __float22half2_rn(reg_y_2); }
  if ((idx + 6)  < N * K) { HALF2(y[idx + 6])  = __float22half2_rn(reg_y_3); }
}

template<const int NUM_THREADS=256>
__global__ void rms_norm_f16_f32_kernel(half* x, half* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = bid * blockDim.x + threadIdx.x;
  const float epsilon = 1e-5f;

  __shared__ float s_variance; // shared within block
  float value = (idx < N * K) ? __half2float(x[idx]) : 0.0f; // load once only
  float variance = value * value;
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / ((float) K + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  if (idx < N * K) {
    y[idx] = __float2half((value * s_variance) * g);
  }
}

template<const int NUM_THREADS=256>
__global__ void rms_norm_f16x8_pack_f16_kernel(half* x, half* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 8;
  const half epsilon = __float2half(1e-5f);
  const half g_      = __float2half(g);
  const half K_      = __int2half_rn(K);
  const half z_      = __float2half(0.0f);

  __shared__ half s_variance; // shared within block
  // temporary register(memory), .local space in ptx, addressable
  half pack_x[8], pack_y[8]; // 8x16 bits=128 bits.
  // reinterpret as float4 and load 128 bits in 1 memory issue.
  LDST128BITS(pack_x[0]) = LDST128BITS(x[idx]); // load 128 bits

  half variance = z_;
  #pragma unroll
  for (int i = 0; i < 8; ++i) {
    variance += ((idx + i) < N * K ? pack_x[i] * pack_x[i] : z_);
  }
  variance = block_reduce_sum_f16_f16<NUM_THREADS>(variance);
  if (tid == 0) s_variance = hrsqrt(variance / (K_ + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 

  #pragma unroll
  for (int i = 0; i < 8; ++i) {
    pack_y[i] = pack_x[i] * s_variance * g_;
  }
  // reinterpret as float4 and store 128 bits in 1 memory issue.
  if ((idx + 7) < N * K) { LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]); }
  // TODO: support non 8-multiple K here
}

template<const int NUM_THREADS=256>
__global__ void rms_norm_f16x8_pack_f32_kernel(half* x, half* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 8;
  const float epsilon = 1e-5f;
  __shared__ float s_variance; // shared within block
  // temporary register(memory), .local space in ptx, addressable
  half pack_x[8], pack_y[8]; // 8x16 bits=128 bits.
  // reinterpret as float4 and load 128 bits in 1 memory issue.
  LDST128BITS(pack_x[0]) = LDST128BITS(x[idx]); // load 128 bits

  float variance = 0.0f;
  #pragma unroll
  for (int i = 0; i < 8; ++i) {
    float v = __half2float(pack_x[i]);
    variance += ((idx + i) < N * K ? v * v : 0.0f);
  }
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / ((float) K + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 

  #pragma unroll
  for (int i = 0; i < 8; i += 2) {
    float2 v2 = __half22float2(HALF2(pack_x[i]));
    float2 y2 = {v2.x * s_variance * g, v2.y * s_variance * g};
    HALF2(pack_y[i]) = __float22half2_rn(y2);
  }
  // reinterpret as float4 and store 128 bits in 1 memory issue.
  if ((idx + 7) < N * K) { LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]); }
  // TODO: support non 8-multiple K here
}


// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T1, T2)               \
assert((T1).dim() == (T2).dim());                      \
for (int i = 0; i < (T1).dim(); ++i) {                 \
  if ((T2).size(i) != (T1).size(i)) {                  \
    throw std::runtime_error("Tensor size mismatch!"); \
  }                                                    \
}

#define LANUCH_RMS_NORM_F32_KERNEL(K)      \
rms_norm_f32_kernel<(K)><<<grid, block>>>( \
  reinterpret_cast<float*>(x.data_ptr()),  \
  reinterpret_cast<float*>(y.data_ptr()),  \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F32_KERNEL(N, K)   \
  dim3 block((K));                           \
  dim3 grid((N));                            \     
  switch ((K))                               \
  {                                          \
  case 64:                                   \
    LANUCH_RMS_NORM_F32_KERNEL(64)           \
    break;                                   \
  case 128:                                  \
    LANUCH_RMS_NORM_F32_KERNEL(128)          \
    break;                                   \
  case 256:                                  \
    LANUCH_RMS_NORM_F32_KERNEL(256)          \
    break;                                   \
  case 512:                                  \
    LANUCH_RMS_NORM_F32_KERNEL(512)          \
    break;                                   \
  case 1024:                                 \
    LANUCH_RMS_NORM_F32_KERNEL(1024)         \
    break;                                   \
  default:                                   \
    throw std::runtime_error(                \
      "only support K: 64/128/256/512/1024");\
    break;                                   \
  } 

#define LANUCH_RMS_NORM_F32x4_KERNEL(K)        \
rms_norm_f32x4_kernel<(K)/4><<<grid, block>>>( \
  reinterpret_cast<float*>(x.data_ptr()),      \
  reinterpret_cast<float*>(y.data_ptr()),      \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F32x4_KERNEL(N, K) \
  dim3 block((K)/4);                         \
  dim3 grid((N));                            \     
  switch ((K))                               \
  {                                          \
  case 64:                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(64)         \
    break;                                   \
  case 128:                                  \
    LANUCH_RMS_NORM_F32x4_KERNEL(128)        \
    break;                                   \
  case 256:                                  \
    LANUCH_RMS_NORM_F32x4_KERNEL(256)        \
    break;                                   \
  case 512:                                  \
    LANUCH_RMS_NORM_F32x4_KERNEL(512)        \
    break;                                   \
  case 1024:                                 \
    LANUCH_RMS_NORM_F32x4_KERNEL(1024)       \
    break;                                   \
  case 2048:                                 \
    LANUCH_RMS_NORM_F32x4_KERNEL(2048)       \
    break;                                   \
  case 4096:                                 \
    LANUCH_RMS_NORM_F32x4_KERNEL(4096)       \
    break;                                   \
  default:                                   \
    throw std::runtime_error(                \
      "only support K: 64/.../512/1024*4");  \
    break;                                   \
  } 

void rms_norm_f32(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kFloat32)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kFloat32)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F32_KERNEL(N, K)
}

void rms_norm_f32x4(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kFloat32)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kFloat32)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F32x4_KERNEL(N, K)
}

// fp16
#define LANUCH_RMS_NORM_F16F16_KERNEL(K)         \
rms_norm_f16_f16_kernel<(K)><<<grid, block>>>(   \
  reinterpret_cast<half*>(x.data_ptr()),         \
  reinterpret_cast<half*>(y.data_ptr()),         \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F16F16_KERNEL(N, K)   \
  dim3 block((K));                              \
  dim3 grid((N));                               \
  switch ((K))                                  \
  {                                             \
  case 64:                                      \
    LANUCH_RMS_NORM_F16F16_KERNEL(64)           \
    break;                                      \
  case 128:                                     \
    LANUCH_RMS_NORM_F16F16_KERNEL(128)          \
    break;                                      \
  case 256:                                     \
    LANUCH_RMS_NORM_F16F16_KERNEL(256)          \
    break;                                      \
  case 512:                                     \
    LANUCH_RMS_NORM_F16F16_KERNEL(512)          \
    break;                                      \
  case 1024:                                    \
    LANUCH_RMS_NORM_F16F16_KERNEL(1024)         \
    break;                                      \
  default:                                      \
    throw std::runtime_error(                   \
      "only support K: 64/128/256/512/1024");   \
    break;                                      \
  } 

#define LANUCH_RMS_NORM_F16F32_KERNEL(K)         \
rms_norm_f16_f32_kernel<(K)><<<grid, block>>>(   \
  reinterpret_cast<half*>(x.data_ptr()),         \
  reinterpret_cast<half*>(y.data_ptr()),         \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F16F32_KERNEL(N, K)   \
  dim3 block((K));                              \
  dim3 grid((N));                               \
  switch ((K))                                  \
  {                                             \
  case 64:                                      \
    LANUCH_RMS_NORM_F16F32_KERNEL(64)           \
    break;                                      \
  case 128:                                     \
    LANUCH_RMS_NORM_F16F32_KERNEL(128)          \
    break;                                      \
  case 256:                                     \
    LANUCH_RMS_NORM_F16F32_KERNEL(256)          \
    break;                                      \
  case 512:                                     \
    LANUCH_RMS_NORM_F16F32_KERNEL(512)          \
    break;                                      \
  case 1024:                                    \
    LANUCH_RMS_NORM_F16F32_KERNEL(1024)         \
    break;                                      \
  default:                                      \
    throw std::runtime_error(                   \
      "only support K: 64/128/256/512/1024");   \
    break;                                      \
  } 

#define LANUCH_RMS_NORM_F16x2F16_KERNEL(K)           \
rms_norm_f16x2_f16_kernel<(K)/2><<<grid, block>>>(   \
  reinterpret_cast<half*>(x.data_ptr()),             \
  reinterpret_cast<half*>(y.data_ptr()),             \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F16x2F16_KERNEL(N, K)   \
  dim3 block((K)/2);                              \
  dim3 grid((N));                                 \
  switch ((K))                                    \
  {                                               \
  case 64:                                        \
    LANUCH_RMS_NORM_F16x2F16_KERNEL(64)           \
    break;                                        \
  case 128:                                       \
    LANUCH_RMS_NORM_F16x2F16_KERNEL(128)          \
    break;                                        \
  case 256:                                       \
    LANUCH_RMS_NORM_F16x2F16_KERNEL(256)          \
    break;                                        \
  case 512:                                       \
    LANUCH_RMS_NORM_F16x2F16_KERNEL(512)          \
    break;                                        \
  case 1024:                                      \
    LANUCH_RMS_NORM_F16x2F16_KERNEL(1024)         \
    break;                                        \
  case 2048:                                      \
    LANUCH_RMS_NORM_F16x2F16_KERNEL(2048)         \
    break;                                        \
  default:                                        \
    throw std::runtime_error(                     \
      "only support K: 64/128/.../1024*2");       \
    break;                                        \
  } 

#define LANUCH_RMS_NORM_F16x8F16_KERNEL(K)           \
rms_norm_f16x8_f16_kernel<(K)/8><<<grid, block>>>(   \
  reinterpret_cast<half*>(x.data_ptr()),             \
  reinterpret_cast<half*>(y.data_ptr()),             \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F16x8F16_KERNEL(N, K)   \
  dim3 block((K)/8);                              \
  dim3 grid((N));                                 \
  switch ((K))                                    \
  {                                               \
  case 64:                                        \
    LANUCH_RMS_NORM_F16x8F16_KERNEL(64)           \
    break;                                        \
  case 128:                                       \
    LANUCH_RMS_NORM_F16x8F16_KERNEL(128)          \
    break;                                        \
  case 256:                                       \
    LANUCH_RMS_NORM_F16x8F16_KERNEL(256)          \
    break;                                        \
  case 512:                                       \
    LANUCH_RMS_NORM_F16x8F16_KERNEL(512)          \
    break;                                        \
  case 1024:                                      \
    LANUCH_RMS_NORM_F16x8F16_KERNEL(1024)         \
    break;                                        \
  case 2048:                                      \
    LANUCH_RMS_NORM_F16x8F16_KERNEL(2048)         \
    break;                                        \
  case 4096:                                      \
    LANUCH_RMS_NORM_F16x8F16_KERNEL(4096)         \
    break;                                        \
  case 8192:                                      \
    LANUCH_RMS_NORM_F16x8F16_KERNEL(8192)         \
    break;                                        \
  default:                                        \
    throw std::runtime_error(                     \
      "only support K: 64/128/.../1024*8");       \
    break;                                        \
  } 

#define LANUCH_RMS_NORM_F16x8F32_KERNEL(K)           \
rms_norm_f16x8_f16_kernel<(K)/8><<<grid, block>>>(   \
  reinterpret_cast<half*>(x.data_ptr()),             \
  reinterpret_cast<half*>(y.data_ptr()),             \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F16x8F32_KERNEL(N, K)   \
  dim3 block((K)/8);                              \
  dim3 grid((N));                                 \
  switch ((K))                                    \
  {                                               \
  case 64:                                        \
    LANUCH_RMS_NORM_F16x8F32_KERNEL(64)           \
    break;                                        \
  case 128:                                       \
    LANUCH_RMS_NORM_F16x8F32_KERNEL(128)          \
    break;                                        \
  case 256:                                       \
    LANUCH_RMS_NORM_F16x8F32_KERNEL(256)          \
    break;                                        \
  case 512:                                       \
    LANUCH_RMS_NORM_F16x8F32_KERNEL(512)          \
    break;                                        \
  case 1024:                                      \
    LANUCH_RMS_NORM_F16x8F32_KERNEL(1024)         \
    break;                                        \
  case 2048:                                      \
    LANUCH_RMS_NORM_F16x8F32_KERNEL(2048)         \
    break;                                        \
  case 4096:                                      \
    LANUCH_RMS_NORM_F16x8F32_KERNEL(4096)         \
    break;                                        \
  case 8192:                                      \
    LANUCH_RMS_NORM_F16x8F32_KERNEL(8192)         \
    break;                                        \
  default:                                        \
    throw std::runtime_error(                     \
      "only support K: 64/128/.../1024*8");       \
    break;                                        \
  } 

#define LANUCH_RMS_NORM_F16x8_PACK_F16_KERNEL(K)        \
rms_norm_f16x8_pack_f16_kernel<(K)/8><<<grid, block>>>( \
  reinterpret_cast<half*>(x.data_ptr()),                \
  reinterpret_cast<half*>(y.data_ptr()),                \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F16x8_PACK_F16_KERNEL(N, K) \
  dim3 block((K)/8);                                  \
  dim3 grid((N));                                     \
  switch ((K))                                        \
  {                                                   \
  case 64:                                            \
    LANUCH_RMS_NORM_F16x8_PACK_F16_KERNEL(64)         \
    break;                                            \
  case 128:                                           \
    LANUCH_RMS_NORM_F16x8_PACK_F16_KERNEL(128)        \
    break;                                            \
  case 256:                                           \
    LANUCH_RMS_NORM_F16x8_PACK_F16_KERNEL(256)        \
    break;                                            \
  case 512:                                           \
    LANUCH_RMS_NORM_F16x8_PACK_F16_KERNEL(512)        \
    break;                                            \
  case 1024:                                          \
    LANUCH_RMS_NORM_F16x8_PACK_F16_KERNEL(1024)       \
    break;                                            \
  case 2048:                                          \
    LANUCH_RMS_NORM_F16x8_PACK_F16_KERNEL(2048)       \
    break;                                            \
  case 4096:                                          \
    LANUCH_RMS_NORM_F16x8_PACK_F16_KERNEL(4096)       \
    break;                                            \
  case 8192:                                          \
    LANUCH_RMS_NORM_F16x8_PACK_F16_KERNEL(8192)       \
    break;                                            \
  default:                                            \
    throw std::runtime_error(                         \
      "only support K: 64/128/.../1024*8");           \
    break;                                            \
  } 

#define LANUCH_RMS_NORM_F16x8_PACK_F32_KERNEL(K)        \
rms_norm_f16x8_pack_f32_kernel<(K)/8><<<grid, block>>>( \
  reinterpret_cast<half*>(x.data_ptr()),                \
  reinterpret_cast<half*>(y.data_ptr()),                \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F16x8_PACK_F32_KERNEL(N, K) \
  dim3 block((K)/8);                                  \
  dim3 grid((N));                                     \
  switch ((K))                                        \
  {                                                   \
  case 64:                                            \
    LANUCH_RMS_NORM_F16x8_PACK_F32_KERNEL(64)         \
    break;                                            \
  case 128:                                           \
    LANUCH_RMS_NORM_F16x8_PACK_F32_KERNEL(128)        \
    break;                                            \
  case 256:                                           \
    LANUCH_RMS_NORM_F16x8_PACK_F32_KERNEL(256)        \
    break;                                            \
  case 512:                                           \
    LANUCH_RMS_NORM_F16x8_PACK_F32_KERNEL(512)        \
    break;                                            \
  case 1024:                                          \
    LANUCH_RMS_NORM_F16x8_PACK_F32_KERNEL(1024)       \
    break;                                            \
  case 2048:                                          \
    LANUCH_RMS_NORM_F16x8_PACK_F32_KERNEL(2048)       \
    break;                                            \
  case 4096:                                          \
    LANUCH_RMS_NORM_F16x8_PACK_F32_KERNEL(4096)       \
    break;                                            \
  case 8192:                                          \
    LANUCH_RMS_NORM_F16x8_PACK_F32_KERNEL(8192)       \
    break;                                            \
  default:                                            \
    throw std::runtime_error(                         \
      "only support K: 64/128/.../1024*8");           \
    break;                                            \
  } 

void rms_norm_f16_f16(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F16F16_KERNEL(N, K)
}

void rms_norm_f16x2_f16(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F16x2F16_KERNEL(N, K)
}

void rms_norm_f16x8_f16(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F16x8F16_KERNEL(N, K)
}

void rms_norm_f16x8_f32(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F16x8F32_KERNEL(N, K)
}

void rms_norm_f16_f32(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F16F32_KERNEL(N, K)
}

// pack
void rms_norm_f16x8_pack_f16(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F16x8_PACK_F16_KERNEL(N, K)
}

void rms_norm_f16x8_pack_f32(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F16x8_PACK_F32_KERNEL(N, K)
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f32)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f32x4)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f16_f16)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f16x2_f16)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f16x8_f16)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f16x8_pack_f16)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f16x8_f32)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f16x8_pack_f32)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f16_f32)
}
