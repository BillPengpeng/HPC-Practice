#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])

// -------------------------------------- FP32 -------------------------------------- 
// Warp Reduce Sum
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

// Block reduce sum/max/min device helper for Layer/RMS Norm/Softmax etc.
// grid 1D block 1D, grid(N/256), block(256)
template<const int NUM_THREADS=256>
__device__ float block_reduce_sum_f32(float val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];
  
  val = warp_reduce_sum_f32<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val;
  __syncthreads();
  val = (lane < NUM_WARPS) ? shared[lane] : 0.0f;
  val = warp_reduce_sum_f32<NUM_WARPS>(val);
  return val;
}

// Layer Norm: x: NxK(K=256<1024), y': NxK, y'=x-mean(x)/std(x) each row
// mean(x) = sum(x)/K, 1/std(x) = rsqrtf( sum( (x-mean(x))^2 )/K ) each row
// grid(N*K/K), block(K<1024) N=batch_size*seq_len, K=hidden_size
// y=y'*g + b (g: scale, b: bias)
template<const int NUM_THREADS=256>
__global__ void layer_norm_f32_kernel(float* x, float* y, float g, float b, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = bid * blockDim.x + threadIdx.x;
  const float epsilon = 1e-5f;

  __shared__ float s_mean; // shared within block
  __shared__ float s_variance; // shared within block
  float value = (idx < N * K) ? x[idx] : 0.0f; // load once only
  float sum = block_reduce_sum_f32<NUM_THREADS>(value);
  if (tid == 0) s_mean = sum / (float) K;
  // wait for s_mean in shared memory to be ready for all threads
  __syncthreads();
  float variance = (value - s_mean) * (value - s_mean);
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / ((float) K + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads();
  if (idx < N * K) y[idx] = ((value - s_mean) * s_variance) * g + b;
}

// Layer Norm Vec4: x: NxK(K=256<1024), y': NxK, y'=x-mean(x)/std(x) each row
// mean(x) = sum(x)/K, 1/std(x) = rsqrtf( sum( (x-mean(x))^2 )/K ) each row
// grid(N*K/K), block(K/4<1024) N=batch_size*seq_len, K=hidden_size
// y=y'*g + b (g: scale, b: bias)
template<const int NUM_THREADS=256/4>
__global__ void layer_norm_f32x4_kernel(float* x, float* y, float g, float b, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 4;
  const float epsilon = 1e-5f;

  __shared__ float s_mean; // shared within block
  __shared__ float s_variance; // shared within block
  float4 reg_x = FLOAT4(x[idx]);
  float value = (idx < N * K) ? (reg_x.x + reg_x.y 
                               + reg_x.z + reg_x.w) : 0.0f;
  float sum = block_reduce_sum_f32<NUM_THREADS>(value);
  if (tid == 0) s_mean = sum / (float) K;
  // wait for s_mean in shared memory to be ready for all threads
  __syncthreads();
  float4 reg_x_hat;
  reg_x_hat.x = reg_x.x - s_mean;
  reg_x_hat.y = reg_x.y - s_mean;
  reg_x_hat.z = reg_x.z - s_mean;
  reg_x_hat.w = reg_x.w - s_mean;
  float variance = reg_x_hat.x * reg_x_hat.x + reg_x_hat.y * reg_x_hat.y 
                 + reg_x_hat.z * reg_x_hat.z + reg_x_hat.w * reg_x_hat.w;
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / ((float) K + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads();
  float4 reg_y;
  reg_y.x = reg_x_hat.x * s_variance * g + b;
  reg_y.y = reg_x_hat.y * s_variance * g + b;
  reg_y.z = reg_x_hat.z * s_variance * g + b;
  reg_y.w = reg_x_hat.w * s_variance * g + b;
  if (idx < N * K) FLOAT4(y[idx]) = reg_y;
}

// -------------------------------------- FP16 -------------------------------------- 
// Warp Reduce Sum: Half
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ half warp_reduce_sum_f16_f16(half val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    // val = __hadd(val, __shfl_xor_sync(0xffffffff, val, mask));
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f16_f32(half val) {
  float val_f32 = __half2float(val);
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val_f32 += __shfl_xor_sync(0xffffffff, val_f32, mask);
  }
  return val_f32;
}

template<const int NUM_THREADS=256>
__device__ half block_reduce_sum_f16_f16(half val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ half shared[NUM_WARPS];
  // reduce using half dtype within warps
  val = warp_reduce_sum_f16_f16<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val;
  __syncthreads();
  val = (lane < NUM_WARPS) ? shared[lane] : __float2half(0.0f);
  val = warp_reduce_sum_f16_f16<NUM_WARPS>(val);
  return val; // half
}

template<const int NUM_THREADS=256>
__device__ float block_reduce_sum_f16_f32(half val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];
  // reduce using float dtype within warps
  float val_f32 = warp_reduce_sum_f16_f32<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val_f32;
  __syncthreads();
  val_f32 = (lane < NUM_WARPS) ? shared[lane] : 0.0f;
  val_f32 = warp_reduce_sum_f32<NUM_WARPS>(val_f32);
  return val_f32; // float
}

template<const int NUM_THREADS=256>
__global__ void layer_norm_f16_f16_kernel(half* x, half* y, float g, float b, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = bid * blockDim.x + threadIdx.x;
  const half epsilon = __float2half(1e-5f);
  const half g_      = __float2half(g);
  const half b_      = __float2half(b);
  const half K_      = __int2half_rn(K);

  __shared__ half s_mean; // shared within block
  __shared__ half s_variance; // shared within block
  half value = (idx < N * K) ? x[idx] : __float2half(0.0f); // load once only
  half sum = block_reduce_sum_f16_f16<NUM_THREADS>(value);
  if (tid == 0) s_mean = sum / K_;
  // wait for s_mean in shared memory to be ready for all threads
  __syncthreads();
  half variance = (value - s_mean) * (value - s_mean);
  variance = block_reduce_sum_f16_f16<NUM_THREADS>(variance);
  if (tid == 0) s_variance = hrsqrt(variance / (K_ + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads();
  if (idx < N * K) { 
    y[idx] = __hfma((value - s_mean) * s_variance, g_, b_);
    // y[idx] = ((value - s_mean) * s_variance) * g_ + b_; 
  }
}

template<const int NUM_THREADS=256>
__global__ void layer_norm_f16x2_f16_kernel(half* x, half* y, float g, float b, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 2;
  const half epsilon = __float2half(1e-5f);
  const half g_      = __float2half(g);
  const half b_      = __float2half(b);
  const half K_      = __int2half_rn(K);

  __shared__ half s_mean; // shared within block
  __shared__ half s_variance; // shared within block
  half2 reg_x = HALF2(x[idx]);
  half value = (idx < N * K) ? (reg_x.x + reg_x.y) : __float2half(0.0f);
  half sum = block_reduce_sum_f16_f16<NUM_THREADS>(value);
  if (tid == 0) s_mean = sum / K_;
  // wait for s_mean in shared memory to be ready for all threads
  __syncthreads();
  half2 reg_x_hat;
  reg_x_hat.x = reg_x.x - s_mean;
  reg_x_hat.y = reg_x.y - s_mean;
  half variance = reg_x_hat.x * reg_x_hat.x + reg_x_hat.y * reg_x_hat.y;
  variance = block_reduce_sum_f16_f16<NUM_THREADS>(variance);
  if (tid == 0) s_variance = hrsqrt(variance / (K_ + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads();
  if (idx < N * K) { 
    half2 reg_y;
    reg_y.x = __hfma(reg_x_hat.x * s_variance, g_, b_);
    reg_y.y = __hfma(reg_x_hat.y * s_variance, g_, b_);
    HALF2(y[idx]) = reg_y;
  }
}

#define HALF2_SUM(reg, i) \
  (((idx + (i)) < N * K) ? ((reg).x + (reg).y) : __float2half(0.0f))

#define HALF2_SUB(reg_y, reg_x) \
  (reg_y).x = (reg_x).x - s_mean; (reg_y).y = (reg_x).y - s_mean;

#define HALF2_VARIANCE(reg, i) \
  (((idx + (i)) < N * K) ? ((reg).x * (reg).x + (reg).y * (reg).y) : __float2half(0.0f))

#define HALF2_LAYER_NORM(reg_y, reg_x, g_, b_)        \
  (reg_y).x = __hfma((reg_x).x * s_variance, g_, b_); \
  (reg_y).y = __hfma((reg_x).y * s_variance, g_, b_);

template<const int NUM_THREADS=256>
__global__ void layer_norm_f16x8_f16_kernel(half* x, half* y, float g, float b, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 8;
  const half epsilon = __float2half(1e-5f);
  const half g_      = __float2half(g);
  const half b_      = __float2half(b);
  const half K_      = __int2half_rn(K);

  __shared__ half s_mean; // shared within block
  __shared__ half s_variance; // shared within block
  half2 reg_x_0 = HALF2(x[idx + 0]);
  half2 reg_x_1 = HALF2(x[idx + 2]);
  half2 reg_x_2 = HALF2(x[idx + 4]);
  half2 reg_x_3 = HALF2(x[idx + 6]);

  half value = HALF2_SUM(reg_x_0, 0);
  value     += HALF2_SUM(reg_x_1, 2);
  value     += HALF2_SUM(reg_x_2, 4);
  value     += HALF2_SUM(reg_x_3, 6);

  half sum = block_reduce_sum_f16_f16<NUM_THREADS>(value);
  if (tid == 0) s_mean = sum / K_;
  // wait for s_mean in shared memory to be ready for all threads
  __syncthreads();
  // manual unroll
  half2 reg_x_hat_0, reg_x_hat_1, reg_x_hat_2, reg_x_hat_3;
  HALF2_SUB(reg_x_hat_0, reg_x_0);
  HALF2_SUB(reg_x_hat_1, reg_x_1);
  HALF2_SUB(reg_x_hat_2, reg_x_2);
  HALF2_SUB(reg_x_hat_3, reg_x_3);

  half variance = HALF2_VARIANCE(reg_x_hat_0, 0);
  variance     += HALF2_VARIANCE(reg_x_hat_1, 2);
  variance     += HALF2_VARIANCE(reg_x_hat_2, 4);
  variance     += HALF2_VARIANCE(reg_x_hat_3, 6);

  variance = block_reduce_sum_f16_f16<NUM_THREADS>(variance);
  if (tid == 0) s_variance = hrsqrt(variance / (K_ + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads();
  // manual unroll
  half2 reg_y_0, reg_y_1, reg_y_2, reg_y_3;
  HALF2_LAYER_NORM(reg_y_0, reg_x_hat_0, g_, b_);
  HALF2_LAYER_NORM(reg_y_1, reg_x_hat_1, g_, b_);
  HALF2_LAYER_NORM(reg_y_2, reg_x_hat_2, g_, b_);
  HALF2_LAYER_NORM(reg_y_3, reg_x_hat_3, g_, b_);
  
  if ((idx + 0) < N * K) { HALF2(y[idx + 0]) = reg_y_0; }
  if ((idx + 2) < N * K) { HALF2(y[idx + 2]) = reg_y_1; }
  if ((idx + 4) < N * K) { HALF2(y[idx + 4]) = reg_y_2; }
  if ((idx + 6) < N * K) { HALF2(y[idx + 6]) = reg_y_3; }
}

template<const int NUM_THREADS=256>
__global__ void layer_norm_f16_f32_kernel(half* x, half* y, float g, float b, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = bid * blockDim.x + threadIdx.x;
  const float epsilon = 1e-5f;

  __shared__ float s_mean; // shared within block
  __shared__ float s_variance; // shared within block
  float value = (idx < N * K) ? __half2float(x[idx]) : 0.0f; // load once only
  float sum = block_reduce_sum_f32<NUM_THREADS>(value);
  if (tid == 0) s_mean = sum / (float) K;
  // wait for s_mean in shared memory to be ready for all threads
  __syncthreads();
  float variance = (value - s_mean) * (value - s_mean);
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / ((float) K + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads();
  if (idx < N * K) {
    // x*y + z -> x'*g + b
    y[idx] = __float2half(
      __fmaf_rn(((value - s_mean) * s_variance), g, b)); 
  }
}

template<const int NUM_THREADS=256>
__global__ void layer_norm_f16x8_pack_f16_kernel(half* x, half* y, float g, float b, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 8;
  const half epsilon = __float2half(1e-5f);
  const half g_      = __float2half(g);
  const half b_      = __float2half(b);
  const half K_      = __int2half_rn(K);
  const half z_      = __float2half(0.0f);

  __shared__ half s_mean; // shared within block
  __shared__ half s_variance; // shared within block
  // temporary register(memory), .local space in ptx, addressable
  half pack_x[8], pack_y[8]; // 8x16 bits=128 bits.
  // reinterpret as float4 and load 128 bits in 1 memory issue.
  LDST128BITS(pack_x[0]) = LDST128BITS(x[idx]); // load 128 bits
  
  half value = z_;
  #pragma unroll
  for (int i = 0; i < 8; ++i) {
    value += ((idx + i) < N * K ? pack_x[i] : z_);
  }
  half sum = block_reduce_sum_f16_f16<NUM_THREADS>(value);
  if (tid == 0) s_mean = sum / K_;
  // wait for s_mean in shared memory to be ready for all threads
  __syncthreads();
  
  half variance = z_;
  #pragma unroll
  for (int i = 0; i < 8; ++i) {
    half v_hat = pack_x[i] - s_mean;
    variance += ((idx + i) < N * K ? v_hat * v_hat : z_);
  }
  variance = block_reduce_sum_f16_f16<NUM_THREADS>(variance);
  if (tid == 0) s_variance = hrsqrt(variance / (K_ + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads();

  #pragma unroll
  for (int i = 0; i < 8; ++i) { 
    // TODO: use __hfma2, __hsub2, __hmul2 here
    pack_y[i] = __hfma((pack_x[i] - s_mean) * s_variance, g_, b_);
  }
  // reinterpret as float4 and store 128 bits in 1 memory issue.
  if ((idx + 7) < N * K) { LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]); }
  // TODO: support non 8-multiple K here
}

template<const int NUM_THREADS=256>
__global__ void layer_norm_f16x8_pack_f32_kernel(half* x, half* y, float g, float b, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 8;
  const float epsilon = 1e-5f;

  __shared__ float s_mean; // shared within block
  __shared__ float s_variance; // shared within block
  // temporary register(memory), .local space in ptx, addressable
  half pack_x[8], pack_y[8]; // 8x16 bits=128 bits.
  // reinterpret as float4 and load 128 bits in 1 memory issue.
  LDST128BITS(pack_x[0]) = LDST128BITS(x[idx]); // load 128 bits

  float value = 0.0f;
  #pragma unroll
  for (int i = 0; i < 8; ++i) {
    value += ((idx + i) < N * K ? __half2float(pack_x[i]) : 0.0f);
  }
  float sum = block_reduce_sum_f32<NUM_THREADS>(value);
  if (tid == 0) s_mean = sum / (float) K;
  // wait for s_mean in shared memory to be ready for all threads
  __syncthreads();

  float variance = 0.0f;
  #pragma unroll
  for (int i = 0; i < 8; ++i) {
    float v_hat = __half2float(pack_x[i]) - s_mean;
    variance += ((idx + i) < N * K ? v_hat * v_hat : 0.0f);
  }
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / ((float) K + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads();
  
  #pragma unroll
  for (int i = 0; i < 8; ++i) {
    pack_y[i] = __float2half(
      __fmaf_rn(((__half2float(pack_x[i]) - s_mean) * s_variance), g, b)
    );
  }
  // reinterpret as float4 and store 128 bits in 1 memory issue.
  if ((idx + 7) < N * K) { LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]); }
  // TODO: support non 8-multiple K here
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T1, T2)               \
assert((T1).dim() == (T2).dim());                      \
for (int i = 0; i < (T1).dim(); ++i) {                 \
  if ((T2).size(i) != (T1).size(i)) {                  \
    throw std::runtime_error("Tensor size mismatch!"); \
  }                                                    \
}

// fp32
#define LANUCH_LAYER_NORM_F32_KERNEL(K)      \
layer_norm_f32_kernel<(K)><<<grid, block>>>( \
  reinterpret_cast<float*>(x.data_ptr()),    \
  reinterpret_cast<float*>(y.data_ptr()),    \
  g, b, N, (K));  

#define DISPATCH_LAYER_NORM_F32_KERNEL(N, K) \
  dim3 block((K));                           \
  dim3 grid((N));                            \
  switch ((K))                               \
  {                                          \
  case 64:                                   \
    LANUCH_LAYER_NORM_F32_KERNEL(64)         \
    break;                                   \
  case 128:                                  \
    LANUCH_LAYER_NORM_F32_KERNEL(128)        \
    break;                                   \
  case 256:                                  \
    LANUCH_LAYER_NORM_F32_KERNEL(256)        \
    break;                                   \
  case 512:                                  \
    LANUCH_LAYER_NORM_F32_KERNEL(512)        \
    break;                                   \
  case 1024:                                 \
    LANUCH_LAYER_NORM_F32_KERNEL(1024)       \
    break;                                   \
  default:                                   \
    throw std::runtime_error(                \
      "only support K: 64/128/256/512/1024");\
    break;                                   \
  } 

#define LANUCH_LAYER_NORM_F32x4_KERNEL(K)        \
layer_norm_f32x4_kernel<(K)/4><<<grid, block>>>( \
  reinterpret_cast<float*>(x.data_ptr()),        \
  reinterpret_cast<float*>(y.data_ptr()),        \
  g, b, N, (K));  

#define DISPATCH_LAYER_NORM_F32x4_KERNEL(N, K) \
  dim3 block((K)/4);                           \
  dim3 grid((N));                              \
  switch ((K))                                 \
  {                                            \
  case 64:                                     \
    LANUCH_LAYER_NORM_F32x4_KERNEL(64)         \
    break;                                     \
  case 128:                                    \
    LANUCH_LAYER_NORM_F32x4_KERNEL(128)        \
    break;                                     \
  case 256:                                    \
    LANUCH_LAYER_NORM_F32x4_KERNEL(256)        \
    break;                                     \
  case 512:                                    \
    LANUCH_LAYER_NORM_F32x4_KERNEL(512)        \
    break;                                     \
  case 1024:                                   \
    LANUCH_LAYER_NORM_F32x4_KERNEL(1024)       \
    break;                                     \
  case 2048:                                   \
    LANUCH_LAYER_NORM_F32x4_KERNEL(2048)       \
    break;                                     \
  case 4096:                                   \
    LANUCH_LAYER_NORM_F32x4_KERNEL(4096)       \
    break;                                     \
  default:                                     \
    throw std::runtime_error(                  \
      "only support K: 64/128/.../1024*4");    \
    break;                                     \
  } 

// fp16
#define LANUCH_LAYER_NORM_F16F16_KERNEL(K)       \
layer_norm_f16_f16_kernel<(K)><<<grid, block>>>( \
  reinterpret_cast<half*>(x.data_ptr()),         \
  reinterpret_cast<half*>(y.data_ptr()),         \
  g, b, N, (K));  

#define DISPATCH_LAYER_NORM_F16F16_KERNEL(N, K) \
  dim3 block((K));                              \
  dim3 grid((N));                               \
  switch ((K))                                  \
  {                                             \
  case 64:                                      \
    LANUCH_LAYER_NORM_F16F16_KERNEL(64)         \
    break;                                      \
  case 128:                                     \
    LANUCH_LAYER_NORM_F16F16_KERNEL(128)        \
    break;                                      \
  case 256:                                     \
    LANUCH_LAYER_NORM_F16F16_KERNEL(256)        \
    break;                                      \
  case 512:                                     \
    LANUCH_LAYER_NORM_F16F16_KERNEL(512)        \
    break;                                      \
  case 1024:                                    \
    LANUCH_LAYER_NORM_F16F16_KERNEL(1024)       \
    break;                                      \
  default:                                      \
    throw std::runtime_error(                   \
      "only support K: 64/128/256/512/1024");   \
    break;                                      \
  } 

#define LANUCH_LAYER_NORM_F16F32_KERNEL(K)       \
layer_norm_f16_f32_kernel<(K)><<<grid, block>>>( \
  reinterpret_cast<half*>(x.data_ptr()),         \
  reinterpret_cast<half*>(y.data_ptr()),         \
  g, b, N, (K));  

#define DISPATCH_LAYER_NORM_F16F32_KERNEL(N, K) \
  dim3 block((K));                              \
  dim3 grid((N));                               \
  switch ((K))                                  \
  {                                             \
  case 64:                                      \
    LANUCH_LAYER_NORM_F16F32_KERNEL(64)         \
    break;                                      \
  case 128:                                     \
    LANUCH_LAYER_NORM_F16F32_KERNEL(128)        \
    break;                                      \
  case 256:                                     \
    LANUCH_LAYER_NORM_F16F32_KERNEL(256)        \
    break;                                      \
  case 512:                                     \
    LANUCH_LAYER_NORM_F16F32_KERNEL(512)        \
    break;                                      \
  case 1024:                                    \
    LANUCH_LAYER_NORM_F16F32_KERNEL(1024)       \
    break;                                      \
  default:                                      \
    throw std::runtime_error(                   \
      "only support K: 64/128/256/512/1024");   \
    break;                                      \
  } 

#define LANUCH_LAYER_NORM_F16x2F16_KERNEL(K)         \
layer_norm_f16x2_f16_kernel<(K)/2><<<grid, block>>>( \
  reinterpret_cast<half*>(x.data_ptr()),             \
  reinterpret_cast<half*>(y.data_ptr()),             \
  g, b, N, (K));  

#define DISPATCH_LAYER_NORM_F16x2F16_KERNEL(N, K) \
  dim3 block((K)/2);                              \
  dim3 grid((N));                                 \
  switch ((K))                                    \
  {                                               \
  case 64:                                        \
    LANUCH_LAYER_NORM_F16x2F16_KERNEL(64)         \
    break;                                        \
  case 128:                                       \
    LANUCH_LAYER_NORM_F16x2F16_KERNEL(128)        \
    break;                                        \
  case 256:                                       \
    LANUCH_LAYER_NORM_F16x2F16_KERNEL(256)        \
    break;                                        \
  case 512:                                       \
    LANUCH_LAYER_NORM_F16x2F16_KERNEL(512)        \
    break;                                        \
  case 1024:                                      \
    LANUCH_LAYER_NORM_F16x2F16_KERNEL(1024)       \
    break;                                        \
  case 2048:                                      \
    LANUCH_LAYER_NORM_F16x2F16_KERNEL(2048)       \
    break;                                        \
  default:                                        \
    throw std::runtime_error(                     \
      "only support K: 64/128/.../1024*2");       \
    break;                                        \
  } 

#define LANUCH_LAYER_NORM_F16x8F16_KERNEL(K)         \
layer_norm_f16x8_f16_kernel<(K)/8><<<grid, block>>>( \
  reinterpret_cast<half*>(x.data_ptr()),             \
  reinterpret_cast<half*>(y.data_ptr()),             \
  g, b, N, (K));  

#define DISPATCH_LAYER_NORM_F16x8F16_KERNEL(N, K) \
  dim3 block((K)/8);                              \
  dim3 grid((N));                                 \
  switch ((K))                                    \
  {                                               \
  case 64:                                        \
    LANUCH_LAYER_NORM_F16x8F16_KERNEL(64)         \
    break;                                        \
  case 128:                                       \
    LANUCH_LAYER_NORM_F16x8F16_KERNEL(128)        \
    break;                                        \
  case 256:                                       \
    LANUCH_LAYER_NORM_F16x8F16_KERNEL(256)        \
    break;                                        \
  case 512:                                       \
    LANUCH_LAYER_NORM_F16x8F16_KERNEL(512)        \
    break;                                        \
  case 1024:                                      \
    LANUCH_LAYER_NORM_F16x8F16_KERNEL(1024)       \
    break;                                        \
  case 2048:                                      \
    LANUCH_LAYER_NORM_F16x8F16_KERNEL(2048)       \
    break;                                        \
  case 4096:                                      \
    LANUCH_LAYER_NORM_F16x8F16_KERNEL(4096)       \
    break;                                        \
  case 8192:                                      \
    LANUCH_LAYER_NORM_F16x8F16_KERNEL(8192)       \
    break;                                        \
  default:                                        \
    throw std::runtime_error(                     \
      "only support K: 64/128/.../1024*8");       \
    break;                                        \
  } 

#define LANUCH_LAYER_NORM_F16x8_PACK_F16_KERNEL(K)        \
layer_norm_f16x8_pack_f16_kernel<(K)/8><<<grid, block>>>( \
  reinterpret_cast<half*>(x.data_ptr()),                  \
  reinterpret_cast<half*>(y.data_ptr()),                  \
  g, b, N, (K));  

#define DISPATCH_LAYER_NORM_F16x8_PACK_F16_KERNEL(N, K) \
  dim3 block((K)/8);                                    \
  dim3 grid((N));                                       \
  switch ((K))                                          \
  {                                                     \
  case 64:                                              \
    LANUCH_LAYER_NORM_F16x8_PACK_F16_KERNEL(64)         \
    break;                                              \
  case 128:                                             \
    LANUCH_LAYER_NORM_F16x8_PACK_F16_KERNEL(128)        \
    break;                                              \
  case 256:                                             \
    LANUCH_LAYER_NORM_F16x8_PACK_F16_KERNEL(256)        \
    break;                                              \
  case 512:                                             \
    LANUCH_LAYER_NORM_F16x8_PACK_F16_KERNEL(512)        \
    break;                                              \
  case 1024:                                            \
    LANUCH_LAYER_NORM_F16x8_PACK_F16_KERNEL(1024)       \
    break;                                              \
  case 2048:                                            \
    LANUCH_LAYER_NORM_F16x8_PACK_F16_KERNEL(2048)       \
    break;                                              \
  case 4096:                                            \
    LANUCH_LAYER_NORM_F16x8_PACK_F16_KERNEL(4096)       \
    break;                                              \
  case 8192:                                            \
    LANUCH_LAYER_NORM_F16x8_PACK_F16_KERNEL(8192)       \
    break;                                              \
  default:                                              \
    throw std::runtime_error(                           \
      "only support K: 64/128/.../1024*8");             \
    break;                                              \
  } 

#define LANUCH_LAYER_NORM_F16x8_PACK_F32_KERNEL(K)        \
layer_norm_f16x8_pack_f32_kernel<(K)/8><<<grid, block>>>( \
  reinterpret_cast<half*>(x.data_ptr()),                  \
  reinterpret_cast<half*>(y.data_ptr()),                  \
  g, b, N, (K));  

#define DISPATCH_LAYER_NORM_F16x8_PACK_F32_KERNEL(N, K) \
  dim3 block((K)/8);                                    \
  dim3 grid((N));                                       \
  switch ((K))                                          \
  {                                                     \
  case 64:                                              \
    LANUCH_LAYER_NORM_F16x8_PACK_F32_KERNEL(64)         \
    break;                                              \
  case 128:                                             \
    LANUCH_LAYER_NORM_F16x8_PACK_F32_KERNEL(128)        \
    break;                                              \
  case 256:                                             \
    LANUCH_LAYER_NORM_F16x8_PACK_F32_KERNEL(256)        \
    break;                                              \
  case 512:                                             \
    LANUCH_LAYER_NORM_F16x8_PACK_F32_KERNEL(512)        \
    break;                                              \
  case 1024:                                            \
    LANUCH_LAYER_NORM_F16x8_PACK_F32_KERNEL(1024)       \
    break;                                              \
  case 2048:                                            \
    LANUCH_LAYER_NORM_F16x8_PACK_F32_KERNEL(2048)       \
    break;                                              \
  case 4096:                                            \
    LANUCH_LAYER_NORM_F16x8_PACK_F32_KERNEL(4096)       \
    break;                                              \
  case 8192:                                            \
    LANUCH_LAYER_NORM_F16x8_PACK_F32_KERNEL(8192)       \
    break;                                              \
  default:                                              \
    throw std::runtime_error(                           \
      "only support K: 64/128/.../1024*8");             \
    break;                                              \
  } 

void layer_norm_f32(torch::Tensor x, torch::Tensor y, float g, float b) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kFloat32)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kFloat32)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_LAYER_NORM_F32_KERNEL(N, K)
}

void layer_norm_f32x4(torch::Tensor x, torch::Tensor y, float g, float b) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kFloat32)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kFloat32)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_LAYER_NORM_F32x4_KERNEL(N, K)
}

void layer_norm_f16_f16(torch::Tensor x, torch::Tensor y, float g, float b) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_LAYER_NORM_F16F16_KERNEL(N, K)
}

void layer_norm_f16x2_f16(torch::Tensor x, torch::Tensor y, float g, float b) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_LAYER_NORM_F16x2F16_KERNEL(N, K)
}

void layer_norm_f16x8_f16(torch::Tensor x, torch::Tensor y, float g, float b) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_LAYER_NORM_F16x8F16_KERNEL(N, K)
}

void layer_norm_f16x8_pack_f16(torch::Tensor x, torch::Tensor y, float g, float b) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_LAYER_NORM_F16x8_PACK_F16_KERNEL(N, K)
}

void layer_norm_f16x8_pack_f32(torch::Tensor x, torch::Tensor y, float g, float b) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_LAYER_NORM_F16x8_PACK_F32_KERNEL(N, K)
}

void layer_norm_f16_f32(torch::Tensor x, torch::Tensor y, float g, float b) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_LAYER_NORM_F16F32_KERNEL(N, K)
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(layer_norm_f32)
  TORCH_BINDING_COMMON_EXTENSION(layer_norm_f32x4)
  TORCH_BINDING_COMMON_EXTENSION(layer_norm_f16_f16)
  TORCH_BINDING_COMMON_EXTENSION(layer_norm_f16_f32)
  TORCH_BINDING_COMMON_EXTENSION(layer_norm_f16x2_f16)
  TORCH_BINDING_COMMON_EXTENSION(layer_norm_f16x8_f16)
  TORCH_BINDING_COMMON_EXTENSION(layer_norm_f16x8_pack_f16)
  TORCH_BINDING_COMMON_EXTENSION(layer_norm_f16x8_pack_f32)
}

