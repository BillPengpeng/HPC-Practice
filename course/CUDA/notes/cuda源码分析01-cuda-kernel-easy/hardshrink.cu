#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])

// 定义全局 LAMBD 值
#define LAMBD 0.5f

// 定义 CHECK_TORCH_TENSOR_DTYPE 宏
#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
  if (((T).options().dtype() != (th_type))) {              \
    std::cout << "Tensor Info:" << (T).options() << std::endl; \
    throw std::runtime_error("Tensor dtype must be " #th_type); \
  }

// 定义 TORCH_BINDING_COMMON_EXTENSION 宏
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
    m.def(STRINGFY(func), &func, STRINGFY(func));



// HARDSHRINK 计算函数
// -------------------------------------- FP32 --------------------------------------
__device__ __forceinline__ float hardshrink(float x) {
  if (x > LAMBD || x < -LAMBD) {
    return x;
  } else {
    return 0;
  }
}


// -------------------------------------- FP16 --------------------------------------
__device__ __forceinline__ half hardshrink_half(half x) {
  if(x > __float2half(LAMBD) || x < __float2half(-LAMBD)) {
    return x;
  } else {
    return __float2half(0.f);
  }
}


// CUDA 核函数
// -------------------------------------- FP32 --------------------------------------
__global__ void hardshrink_f32_kernel(float* x, float* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = hardshrink(x[idx]);
}

__global__ void hardshrink_f32x4_kernel(float* x, float* y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  if (idx < N) {
    float4 reg_x = FLOAT4(x[idx]);
    float4 reg_y;
    reg_y.x = hardshrink(reg_x.x);
    reg_y.y = hardshrink(reg_x.y);
    reg_y.z = hardshrink(reg_x.z);
    reg_y.w = hardshrink(reg_x.w);
    FLOAT4(y[idx]) = reg_y;
  }
}

// -------------------------------------- FP16 --------------------------------------
__global__ void hardshrink_f16_kernel(half* x, half* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = hardshrink_half(x[idx]);
}


__global__ void hardshrink_f16x2_kernel(half* x, half* y, int N) {
  int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
  if (idx < N) {
    half2 reg_x = HALF2(x[idx]);
    half2 reg_y;
    reg_y.x = hardshrink_half(reg_x.x);
    reg_y.y = hardshrink_half(reg_x.y);
    HALF2(y[idx]) = reg_y;
  }
}



__global__ void hardshrink_f16x8_kernel(half* x, half* y, int N) {
  int idx = 8 * (blockIdx.x * blockDim.x + threadIdx.x);
  half2 reg_x_0 = HALF2(x[idx + 0]);
  half2 reg_x_1 = HALF2(x[idx + 2]);
  half2 reg_x_2 = HALF2(x[idx + 4]);
  half2 reg_x_3 = HALF2(x[idx + 6]);
  half2 reg_y_0, reg_y_1, reg_y_2, reg_y_3;
  reg_y_0.x = hardshrink_half(reg_x_0.x);
  reg_y_0.y = hardshrink_half(reg_x_0.y);
  reg_y_1.x = hardshrink_half(reg_x_1.x);
  reg_y_1.y = hardshrink_half(reg_x_1.y);
  reg_y_2.x = hardshrink_half(reg_x_2.x);
  reg_y_2.y = hardshrink_half(reg_x_2.y);
  reg_y_3.x = hardshrink_half(reg_x_3.x);
  reg_y_3.y = hardshrink_half(reg_x_3.y);
  if ((idx + 0) < N) { HALF2(y[idx + 0]) = reg_y_0; }
  if ((idx + 2) < N) { HALF2(y[idx + 2]) = reg_y_1; }
  if ((idx + 4) < N) { HALF2(y[idx + 4]) = reg_y_2; }
  if ((idx + 6) < N) { HALF2(y[idx + 6]) = reg_y_3; }
}



__global__ void hardshrink_f16x8_pack_kernel(half* x, half* y, int N) {
  int idx = 8 * (blockIdx.x * blockDim.x + threadIdx.x);
  half pack_x[8], pack_y[8];
  LDST128BITS(pack_x[0]) = LDST128BITS(x[idx]);

  #pragma unroll
  for (int i = 0; i < 8; i++) {
    pack_y[i] = hardshrink_half(pack_x[i]);
  }
  if ((idx + 7) < N) { LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]); }
}

// PyTorch 绑定代码
#define TORCH_BINDING_HARDSHRINK(packed_type, th_type, element_type, n_elements)      \
void hardshrink_##packed_type(torch::Tensor x, torch::Tensor y) {                     \
  CHECK_TORCH_TENSOR_DTYPE(x, (th_type))                                     \
  CHECK_TORCH_TENSOR_DTYPE(y, (th_type))                                     \
  const int ndim = x.dim();                                                  \
  if (ndim != 2) {                                                           \
    int N = 1;                                                             \
    for (int i = 0; i < ndim; ++i) { N *= x.size(i); }                     \
    dim3 block(256 / (n_elements));                                        \
    dim3 grid((N + 256 - 1) / 256);                                        \
    hardshrink_##packed_type##_kernel<<<grid, block>>>(                           \
        reinterpret_cast<element_type*>(x.data_ptr()),                     \
        reinterpret_cast<element_type*>(y.data_ptr()), N);                 \
  } else {                                                                   \
    const int S = x.size(0);                                               \
    const int K = x.size(1);                                               \
    const int N = S * K;                                                   \
    if ((K/(n_elements)) <= 1024) {                                        \
      dim3 block(K/(n_elements));                                        \
      dim3 grid(S);                                                      \
      hardshrink_##packed_type##_kernel<<<grid, block>>>(                       \
        reinterpret_cast<element_type*>(x.data_ptr()),                 \
        reinterpret_cast<element_type*>(y.data_ptr()), N);             \
  } else {                                                               \
    int N = 1;                                                         \
    for (int i = 0; i < ndim; ++i) { N *= x.size(i); }                 \
    dim3 block(256 / (n_elements));                                    \
    dim3 grid((N + 256 - 1) / 256);                                    \
    hardshrink_##packed_type##_kernel<<<grid, block>>>(                       \
    reinterpret_cast<element_type*>(x.data_ptr()),                 \
    reinterpret_cast<element_type*>(y.data_ptr()), N);             \
    }                                                                      \
    }                                                                          \
}

TORCH_BINDING_HARDSHRINK(f32,        torch::kFloat32,    float,    1)
TORCH_BINDING_HARDSHRINK(f32x4,      torch::kFloat32,    float,    4)
TORCH_BINDING_HARDSHRINK(f16,        torch::kHalf,       half,     1)
TORCH_BINDING_HARDSHRINK(f16x2,      torch::kHalf,       half,     2)
TORCH_BINDING_HARDSHRINK(f16x8,      torch::kHalf,       half,     8)
TORCH_BINDING_HARDSHRINK(f16x8_pack, torch::kHalf,       half,     8)

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
TORCH_BINDING_COMMON_EXTENSION(hardshrink_f32)
TORCH_BINDING_COMMON_EXTENSION(hardshrink_f32x4)
TORCH_BINDING_COMMON_EXTENSION(hardshrink_f16)
TORCH_BINDING_COMMON_EXTENSION(hardshrink_f16x2)
TORCH_BINDING_COMMON_EXTENSION(hardshrink_f16x8)
TORCH_BINDING_COMMON_EXTENSION(hardshrink_f16x8_pack)
}