#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])
#define MAX_EXP_F32  88.3762626647949f
#define MIN_EXP_F32 -88.3762626647949f
#define MAX_EXP_F16 __float2half(11.089866488461016f)
#define MIN_EXP_F16 __float2half(-9.704060527839234f)
#define SQRT_2_PI M_SQRT2 * M_2_SQRTPI * 0.5f
#define HALF_1 __float2half(1.0f)
#define HALF_2 __float2half(2.0f)
#define HALF_DIV2 __float2half(0.5f)
// to clear the error among self defined gelu and pytorch gelu. Calculate $\sqrt{\frac{\pi}{2}}$ by $\sqrt{2 * \pi} / 2$
#define HALF_SQRT_2_PI __float2half(M_SQRT2) * __float2half(M_2_SQRTPI) * HALF_DIV2
#define HALF_V_APP __float2half(0.044715f)

#define HALF_GELU_OPS gelu_tanh_approximate
#define GELU_OPS gelu_tanh_approximate

// There is no half presicion operation like sinh, cosh, tanh. [Half Math Functions](https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH____HALF__FUNCTIONS.html#group__CUDA__MATH____HALF__FUNCTIONS)
// $$ tanh(x) = \frac{exp^{2x} - 1}{exp^{2x} + 1}$$ 
// But ops above will introduce error. 
// pytorch transform type while do tanh operator which include in the [pytorch/c10/util/BFloat16-math.h](https://github.com/pytorch/pytorch/blob/main/c10/util/BFloat16-math.h)
__inline__ __device__ half gelu_tanh_approximate(half x){
  half x_cube = x * x * x;
  // compute mid value : inner = 0.7978845608 * (x + 0.044715 * x * x * x)
  half inner = HALF_SQRT_2_PI * (x + HALF_V_APP * x_cube);
  // compute tanh
  return HALF_DIV2 * x * (HALF_1 + ((hexp(inner * HALF_2) - HALF_1) / (hexp(inner * HALF_2) + HALF_1))); 
}

__inline__ __device__ float gelu_tanh_approximate(float x){
  return 0.5f * x * (1.0f + tanhf(SQRT_2_PI * (x + 0.044715f * x * x * x)));
}

__inline__ __device__ float gelu_none_approximate(float x){
  return x * 0.5 * (1 + erff(x  * M_SQRT1_2));
}

// -------------------------------------- FP32 -------------------------------------- 
// GELU tanh approximate: x, y:x 0.5 * x * (1.0 + tanh(0.7978845608 * x * (1.0 + 0.044715 * x * x)))
// grid(N/256), block(K=256) 
__global__ void gelu_f32_kernel(float* x, float* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    float v = fminf(fmaxf(x[idx], MIN_EXP_F32), MAX_EXP_F32);
    y[idx] = GELU_OPS(v);
  }
}

// GELU tanh approximate; Vec4
// grid(N/256), block(256/4)
__global__ void gelu_f32x4_kernel(float* x, float* y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  float4 reg_x = FLOAT4(x[idx]);
  float4 reg_y;
    
  reg_x.x = fminf(fmaxf(reg_x.x, MIN_EXP_F32), MAX_EXP_F32);
  reg_x.y = fminf(fmaxf(reg_x.y, MIN_EXP_F32), MAX_EXP_F32);
  reg_x.z = fminf(fmaxf(reg_x.z, MIN_EXP_F32), MAX_EXP_F32);
  reg_x.w = fminf(fmaxf(reg_x.w, MIN_EXP_F32), MAX_EXP_F32);

  reg_y.x = GELU_OPS(reg_x.x);
  reg_y.y = GELU_OPS(reg_x.y);
  reg_y.z = GELU_OPS(reg_x.z);
  reg_y.w = GELU_OPS(reg_x.w);

  if ((idx + 0) < N) { FLOAT4(y[idx]) = reg_y; }
}

// -------------------------------------- FP16 -------------------------------------- 
// GELU approximate: x, y:x 0.5 * x * (1.0 + tanh(0.7978845608 (x + 0.044715 * x * x * x))) Vec4
__global__ void gelu_f16_kernel(half* x, half* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    half v = x[idx];
    v = __hmin(__hmax(v, MIN_EXP_F16), MAX_EXP_F16);
    
    y[idx] = HALF_GELU_OPS(v);
  }
}

__global__ void gelu_f16x2_kernel(half* x, half* y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 2;

  half2 reg_x = HALF2(x[idx]);
  half2 reg_y;
  reg_x.x = __hmin(__hmax(reg_x.x, MIN_EXP_F16), MAX_EXP_F16);
  reg_x.y = __hmin(__hmax(reg_x.y, MIN_EXP_F16), MAX_EXP_F16);

  reg_y.x = HALF_GELU_OPS(reg_x.x);
  reg_y.y = HALF_GELU_OPS(reg_x.y);
  if ((idx + 0) < N) { HALF2(y[idx]) = reg_y; }
}

// unpack f16x8
__global__ void gelu_f16x8_kernel(half* x, half* y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 8;

  half2 reg_x_0 = HALF2(x[idx + 0]);
  half2 reg_x_1 = HALF2(x[idx + 2]);
  half2 reg_x_2 = HALF2(x[idx + 4]);
  half2 reg_x_3 = HALF2(x[idx + 6]);

  reg_x_0.x = __hmin(__hmax(reg_x_0.x, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_0.y = __hmin(__hmax(reg_x_0.y, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_1.x = __hmin(__hmax(reg_x_1.x, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_1.y = __hmin(__hmax(reg_x_1.y, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_2.x = __hmin(__hmax(reg_x_2.x, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_2.y = __hmin(__hmax(reg_x_2.y, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_3.x = __hmin(__hmax(reg_x_3.x, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_3.y = __hmin(__hmax(reg_x_3.y, MIN_EXP_F16), MAX_EXP_F16);

  half2 reg_y_0, reg_y_1, reg_y_2, reg_y_3;

  reg_x_0.x = HALF_GELU_OPS(reg_x_0.x);
  reg_x_0.y = HALF_GELU_OPS(reg_x_0.y);
  reg_x_1.x = HALF_GELU_OPS(reg_x_1.x);
  reg_x_1.y = HALF_GELU_OPS(reg_x_1.y);
  reg_x_2.x = HALF_GELU_OPS(reg_x_2.x);
  reg_x_2.y = HALF_GELU_OPS(reg_x_2.y);
  reg_x_3.x = HALF_GELU_OPS(reg_x_3.x);
  reg_x_3.y = HALF_GELU_OPS(reg_x_3.y);

  if ((idx + 0) < N) { HALF2(y[idx + 0]) = reg_x_0; }
  if ((idx + 2) < N) { HALF2(y[idx + 2]) = reg_x_1; }
  if ((idx + 4) < N) { HALF2(y[idx + 4]) = reg_x_2; }
  if ((idx + 6) < N) { HALF2(y[idx + 6]) = reg_x_3; }
}

// pack f16x8
__global__ void gelu_f16x8_pack_kernel(half* x, half* y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
  
  // temporary register(memory), .local space in ptx, addressable
  half pack_x[8], pack_y[8]; // 8x16 bits=128 bits.
  // reinterpret as float4 and load 128 bits in 1 memory issue.
  LDST128BITS(pack_x[0]) = LDST128BITS(x[idx]); // load 128 bits
  
  #pragma unroll
  for (int i = 0; i < 8; ++i) {
    half v = __hmin(__hmax(pack_x[i], MIN_EXP_F16), MAX_EXP_F16);
    pack_y[i] = HALF_GELU_OPS(v);
  }
  // reinterpret as float4 and store 128 bits in 1 memory issue.
  if ((idx + 7) < N) { LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]); }
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define TORCH_BINDING_GELU(packed_type, th_type, element_type, n_elements)       \
void gelu_##packed_type(torch::Tensor x, torch::Tensor y) {                      \
  CHECK_TORCH_TENSOR_DTYPE(x, (th_type))                                         \
  CHECK_TORCH_TENSOR_DTYPE(y, (th_type))                                         \
  const int ndim = x.dim();                                                      \
  if (ndim != 2) {                                                               \
    int N = 1;                                                                   \
    for (int i = 0; i < ndim; ++i) { N *= x.size(i); }                           \
    dim3 block(256 / (n_elements));                                              \
    dim3 grid((N + 256 - 1) / 256);                                              \
    gelu_##packed_type##_kernel<<<grid, block>>>(                                \
      reinterpret_cast<element_type*>(x.data_ptr()),                             \
      reinterpret_cast<element_type*>(y.data_ptr()), N);                         \
  } else {                                                                       \
    const int S = x.size(0);                                                     \
    const int K = x.size(1);                                                     \
    const int N = S * K;                                                         \
    if ((K/(n_elements)) <= 1024) {                                              \
      dim3 block(K/(n_elements));                                                \
      dim3 grid(S);                                                              \
      gelu_##packed_type##_kernel<<<grid, block>>>(                              \
        reinterpret_cast<element_type*>(x.data_ptr()),                           \
        reinterpret_cast<element_type*>(y.data_ptr()), N);                       \
    } else {                                                                     \
      int N = 1;                                                                 \
      for (int i = 0; i < ndim; ++i) { N *= x.size(i); }                         \
      dim3 block(256 / (n_elements));                                            \
      dim3 grid((N + 256 - 1) / 256);                                            \
      gelu_##packed_type##_kernel<<<grid, block>>>(                              \
        reinterpret_cast<element_type*>(x.data_ptr()),                           \
        reinterpret_cast<element_type*>(y.data_ptr()), N);                       \
    }                                                                            \
  }                                                                              \
}


TORCH_BINDING_GELU(f32,        torch::kFloat32,    float,    1)
TORCH_BINDING_GELU(f32x4,      torch::kFloat32,    float,    4)
TORCH_BINDING_GELU(f16,        torch::kHalf,       half,     1)
TORCH_BINDING_GELU(f16x2,      torch::kHalf,       half,     2)
TORCH_BINDING_GELU(f16x8,      torch::kHalf,       half,     8)
TORCH_BINDING_GELU(f16x8_pack, torch::kHalf,       half,     8)

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(gelu_f32)
  TORCH_BINDING_COMMON_EXTENSION(gelu_f32x4)
  TORCH_BINDING_COMMON_EXTENSION(gelu_f16)
  TORCH_BINDING_COMMON_EXTENSION(gelu_f16x2)
  TORCH_BINDING_COMMON_EXTENSION(gelu_f16x8)
  TORCH_BINDING_COMMON_EXTENSION(gelu_f16x8_pack)
}
