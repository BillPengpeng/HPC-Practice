#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])

__global__ void embedding_f32_kernel(const int *idx, float *weight, float *output, int n, int emb_size)
{
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int tid = bx * blockDim.x + tx;
  int offset = idx[bx] * emb_size;
  output[bx * emb_size + tx] = weight[offset + tx];
}

__global__ void embedding_f32x4_kernel(const int *idx, float *weight, float *output, int n, int emb_size)
{
  int tx = threadIdx.x * 4;
  int bx = blockIdx.x;
  int offset = idx[bx] * emb_size;
  output[bx * emb_size + tx] = weight[offset + tx];
  output[bx * emb_size + tx + 1] = weight[offset + tx + 1];
  output[bx * emb_size + tx + 2] = weight[offset + tx + 2];
  output[bx * emb_size + tx + 3] = weight[offset + tx + 3];
}

__global__ void embedding_f32x4_pack_kernel(const int *idx, float *weight, float *output, int n, int emb_size)
{
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int tid = bx * blockDim.x + tx;
  int offset = idx[bx] * emb_size;
  LDST128BITS(output[bx * emb_size + 4 * tx]) = LDST128BITS(weight[offset + 4 * tx]);
}

__global__ void embedding_f16_kernel(const int *idx, half *weight, half *output, int n, int emb_size)
{
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int tid = bx * blockDim.x + tx;
  int offset = idx[bx] * emb_size;
  output[bx * emb_size + tx] = weight[offset + tx];
}

__global__ void embedding_f16x8_kernel(const int *idx, half *weight, half *output, int n, int emb_size)
{
  int tx = threadIdx.x * 8;
  int bx = blockIdx.x;
  int offset = idx[bx] * emb_size;
  output[bx * emb_size + tx] = weight[offset + tx];
  output[bx * emb_size + tx + 1] = weight[offset + tx + 1];
  output[bx * emb_size + tx + 2] = weight[offset + tx + 2];
  output[bx * emb_size + tx + 3] = weight[offset + tx + 3];
  output[bx * emb_size + tx + 4] = weight[offset + tx + 4];
  output[bx * emb_size + tx + 5] = weight[offset + tx + 5];
  output[bx * emb_size + tx + 6] = weight[offset + tx + 6];
  output[bx * emb_size + tx + 7] = weight[offset + tx + 7];
}

__global__ void embedding_f16x8_pack_kernel(const int *idx, half *weight, half *output, int n, int emb_size)
{
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int tid = bx * blockDim.x + tx;
  int offset = idx[bx] * emb_size;
  LDST128BITS(output[bx * emb_size + 8 * tx]) = LDST128BITS(weight[offset + 8 * tx]);
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
    m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                       \
    if (((T).options().dtype() != (th_type)))                      \
    {                                                              \
        std::cout << "Tensor Info:" << (T).options() << std::endl; \
        throw std::runtime_error("values must be " #th_type);      \
    }

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)                \
    if (((T).size(0) != (S0)) || ((T).size(1) != (S1)))    \
    {                                                      \
        throw std::runtime_error("Tensor size mismatch!"); \
    }

#define TORCH_BINDING_EMBEDDING(packed_type, th_type, element_type, n_elements) \
    void embedding_##packed_type(                                               \
        torch::Tensor a, torch::Tensor weight, torch::Tensor o)                 \
    {                                                                           \
        CHECK_TORCH_TENSOR_DTYPE(a, (torch::kInt32));                           \
        CHECK_TORCH_TENSOR_DTYPE(weight, (th_type));                            \
        CHECK_TORCH_TENSOR_DTYPE(o, (th_type));                                 \
                                                                                \                        
        const int N = a.size(0);                                                \
        const int emb_size = weight.size(1);                                    \
        dim3 block(emb_size / n_elements);                                      \
        dim3 grid(N);                                                           \
        embedding_##packed_type##_kernel<<<grid, block>>>(                      \
            reinterpret_cast<int *>(a.data_ptr()),                              \
            reinterpret_cast<element_type *>(weight.data_ptr()),                \
            reinterpret_cast<element_type *>(o.data_ptr()), N, emb_size);       \
    }

TORCH_BINDING_EMBEDDING(f32,        torch::kFloat32,  float,  1)
TORCH_BINDING_EMBEDDING(f32x4,      torch::kFloat32,  float,  4)
TORCH_BINDING_EMBEDDING(f32x4_pack, torch::kFloat32,  float,  4)
TORCH_BINDING_EMBEDDING(f16,        torch::kHalf,     half,   1)
TORCH_BINDING_EMBEDDING(f16x8,      torch::kHalf,     half,   8)
TORCH_BINDING_EMBEDDING(f16x8_pack, torch::kHalf,     half,   8)

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    TORCH_BINDING_COMMON_EXTENSION(embedding_f32);
    TORCH_BINDING_COMMON_EXTENSION(embedding_f32x4);
    TORCH_BINDING_COMMON_EXTENSION(embedding_f32x4_pack);
    TORCH_BINDING_COMMON_EXTENSION(embedding_f16);
    TORCH_BINDING_COMMON_EXTENSION(embedding_f16x8);
    TORCH_BINDING_COMMON_EXTENSION(embedding_f16x8_pack);
}
