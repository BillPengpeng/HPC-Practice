#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])

// -------------------------------------- FP32 -------------------------------------- 
// Relu x: N, y: N y=max(0,x)
// grid(N/256), block(K=256) 
__global__ void relu_f32_kernel(float* x, float* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = fmaxf(0.0f, x[idx]);
}

// Relu x: N, y: N y=max(0,x) Vec4
// grid(N/256/4), block(256/4) 
__global__ void relu_f32x4_kernel(float* x, float* y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  if (idx < N) {
    float4 reg_x = FLOAT4(x[idx]);
    float4 reg_y;
    reg_y.x = fmaxf(0.0f, reg_x.x);
    reg_y.y = fmaxf(0.0f, reg_x.y);
    reg_y.z = fmaxf(0.0f, reg_x.z);
    reg_y.w = fmaxf(0.0f, reg_x.w);
    FLOAT4(y[idx]) = reg_y;
  }
}

// -------------------------------------- FP16 -------------------------------------- 
__global__ void relu_f16_kernel(half* x, half* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = __hmax(__float2half(0.0f), x[idx]);
}

__global__ void relu_f16x2_kernel(half* x, half* y, int N) {
  int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
  if (idx < N) {
    half2 reg_x = HALF2(x[idx]);
    half2 reg_y = HALF2(y[idx]);
    reg_y.x = __hmax(__float2half(0.0f), reg_x.x);
    reg_y.y = __hmax(__float2half(0.0f), reg_x.y);
    HALF2(y[idx]) = reg_y;
  }
}

__global__ void relu_f16x8_kernel(half* x, half* y, int N) {
  int idx = 8 * (blockIdx.x * blockDim.x + threadIdx.x);
  half2 reg_x_0 = HALF2(x[idx + 0]);
  half2 reg_x_1 = HALF2(x[idx + 2]);
  half2 reg_x_2 = HALF2(x[idx + 4]);
  half2 reg_x_3 = HALF2(x[idx + 6]);
  half2 reg_y_0, reg_y_1, reg_y_2, reg_y_3;
  reg_y_0.x = __hmax(__float2half(0.0f), reg_x_0.x);
  reg_y_0.y = __hmax(__float2half(0.0f), reg_x_0.y);
  reg_y_1.x = __hmax(__float2half(0.0f), reg_x_1.x);
  reg_y_1.y = __hmax(__float2half(0.0f), reg_x_1.y);
  reg_y_2.x = __hmax(__float2half(0.0f), reg_x_2.x);
  reg_y_2.y = __hmax(__float2half(0.0f), reg_x_2.y);
  reg_y_3.x = __hmax(__float2half(0.0f), reg_x_3.x);
  reg_y_3.y = __hmax(__float2half(0.0f), reg_x_3.y);
  if ((idx + 0) < N) { HALF2(y[idx + 0]) = reg_y_0; }
  if ((idx + 2) < N) { HALF2(y[idx + 2]) = reg_y_1; }
  if ((idx + 4) < N) { HALF2(y[idx + 4]) = reg_y_2; }
  if ((idx + 6) < N) { HALF2(y[idx + 6]) = reg_y_3; }
}

__global__ void relu_f16x8_pack_kernel(half* x, half* y, int N) {
  int idx = 8 * (blockIdx.x * blockDim.x + threadIdx.x);
  const half2 z2 = {__float2half(0.0f), __float2half(0.0f)};
  // temporary register(memory), .local space in ptx, addressable
  half pack_x[8], pack_y[8]; // 8x16 bits=128 bits.
  // reinterpret as float4 and load 128 bits in 1 memory issue.
  LDST128BITS(pack_x[0]) = LDST128BITS(x[idx]); // load 128 bits

  #pragma unroll
  for (int i = 0; i < 8; i += 2) {
    // __hmax2 for half2 x 4
    HALF2(pack_y[i]) = __hmax2(HALF2(pack_x[i]), z2);
  } 
  // reinterpret as float4 and store 128 bits in 1 memory issue.
  if ((idx + 7) < N) { LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]); }
}


// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define TORCH_BINDING_RELU(packed_type, th_type, element_type, n_elements)       \
void relu_##packed_type(torch::Tensor x, torch::Tensor y) {                      \
  CHECK_TORCH_TENSOR_DTYPE(x, (th_type))                                         \
  CHECK_TORCH_TENSOR_DTYPE(y, (th_type))                                         \
  const int ndim = x.dim();                                                      \
  if (ndim != 2) {                                                               \
    int N = 1;                                                                   \
    for (int i = 0; i < ndim; ++i) { N *= x.size(i); }                           \
    dim3 block(256 / (n_elements));                                              \
    dim3 grid((N + 256 - 1) / 256);                                              \
    relu_##packed_type##_kernel<<<grid, block>>>(                                \
      reinterpret_cast<element_type*>(x.data_ptr()),                             \
      reinterpret_cast<element_type*>(y.data_ptr()), N);                         \
  } else {                                                                       \
    const int S = x.size(0);                                                     \
    const int K = x.size(1);                                                     \
    const int N = S * K;                                                         \
    if ((K/(n_elements)) <= 1024) {                                              \
      dim3 block(K/(n_elements));                                                \
      dim3 grid(S);                                                              \
      relu_##packed_type##_kernel<<<grid, block>>>(                              \
        reinterpret_cast<element_type*>(x.data_ptr()),                           \
        reinterpret_cast<element_type*>(y.data_ptr()), N);                       \
    } else {                                                                     \
      int N = 1;                                                                 \
      for (int i = 0; i < ndim; ++i) { N *= x.size(i); }                         \
      dim3 block(256 / (n_elements));                                            \
      dim3 grid((N + 256 - 1) / 256);                                            \
      relu_##packed_type##_kernel<<<grid, block>>>(                              \
        reinterpret_cast<element_type*>(x.data_ptr()),                           \
        reinterpret_cast<element_type*>(y.data_ptr()), N);                       \
    }                                                                            \
  }                                                                              \
}


TORCH_BINDING_RELU(f32,        torch::kFloat32,    float,    1)
TORCH_BINDING_RELU(f32x4,      torch::kFloat32,    float,    4)
TORCH_BINDING_RELU(f16,        torch::kHalf,       half,     1)
TORCH_BINDING_RELU(f16x2,      torch::kHalf,       half,     2)
TORCH_BINDING_RELU(f16x8,      torch::kHalf,       half,     8)
TORCH_BINDING_RELU(f16x8_pack, torch::kHalf,       half,     8)

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(relu_f32)
  TORCH_BINDING_COMMON_EXTENSION(relu_f32x4)
  TORCH_BINDING_COMMON_EXTENSION(relu_f16)
  TORCH_BINDING_COMMON_EXTENSION(relu_f16x2)
  TORCH_BINDING_COMMON_EXTENSION(relu_f16x8)
  TORCH_BINDING_COMMON_EXTENSION(relu_f16x8_pack)
}
