#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <tuple>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

// Histogram
// grid(N/256), block(256)
// a: Nx1, y: count histogram, a >= 1
__global__ void histogram_i32_kernel(int* a, int* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) atomicAdd(&(y[a[idx]]), 1);
}

// Histogram + Vec4
// grid(N/256), block(256/4)
// a: Nx1, y: count histogram, a >= 1
__global__ void histogram_i32x4_kernel(int* a, int* y, int N) {
  int idx = 4 * (blockIdx.x * blockDim.x + threadIdx.x);
  if (idx < N) {
    int4 reg_a = INT4(a[idx]);
    atomicAdd(&(y[reg_a.x]), 1);
    atomicAdd(&(y[reg_a.y]), 1);
    atomicAdd(&(y[reg_a.z]), 1);
    atomicAdd(&(y[reg_a.w]), 1);
  }
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T, S0) \
if (((T).size(0) != (S0))) { throw std::runtime_error("Tensor size mismatch!"); }

#define TORCH_BINDING_HIST(packed_type, th_type, element_type, n_elements)       \
torch::Tensor histogram_##packed_type(torch::Tensor a) {                         \
  CHECK_TORCH_TENSOR_DTYPE(a, (th_type))                                         \
  auto options = torch::TensorOptions().dtype(torch::kInt32).device(             \
    torch::kCUDA, 0);                                                            \
  const int N = a.size(0);                                                       \
  std::tuple<torch::Tensor, torch::Tensor> max_a = torch::max(a, 0);             \
  torch::Tensor max_val = std::get<0>(max_a).cpu();                              \
  const int M = max_val.item().to<int>();                                        \
  auto y = torch::zeros({M+1}, options);                                         \
  static const int NUM_THREADS_PER_BLOCK = 256 / (n_elements);                   \
  const int NUM_BLOCKS = (N + 256 - 1) / 256;                                    \
  dim3 block(NUM_THREADS_PER_BLOCK);                                             \
  dim3 grid(NUM_BLOCKS);                                                         \
  histogram_##packed_type##_kernel<<<grid, block>>>(                             \
      reinterpret_cast<element_type*>(a.data_ptr()),                             \
      reinterpret_cast<element_type*>(y.data_ptr()), N);                         \
  return y;                                                                      \
}

TORCH_BINDING_HIST(i32,   torch::kInt32, int, 1)
TORCH_BINDING_HIST(i32x4, torch::kInt32, int, 4)

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(histogram_i32)
  TORCH_BINDING_COMMON_EXTENSION(histogram_i32x4)
}
