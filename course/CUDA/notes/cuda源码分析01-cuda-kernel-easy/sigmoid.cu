#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])
#define MAX_EXP_F32  88.3762626647949f
#define MIN_EXP_F32 -88.3762626647949f
#define MAX_EXP_F16 __float2half(11.089866488461016f)
#define MIN_EXP_F16 __float2half(-9.704060527839234f)


// -------------------------------------- FP32 -------------------------------------- 
// Sigmoid x: N, y: N y=1/(1+exp(-x))
// grid(N/256), block(K=256) 
__global__ void sigmoid_f32_kernel(float* x, float* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    float v = x[idx];
    v = fminf(fmaxf(v, MIN_EXP_F32), MAX_EXP_F32);
    y[idx] = 1.0f / (1.0f + expf(-v));
  }
}

// Sigmoid x: N, y: N y=1/(1+exp(-x)) Vec4
// grid(N/256), block(256/4)
__global__ void sigmoid_f32x4_kernel(float* x, float* y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  float4 reg_x = FLOAT4(x[idx]);
  float4 reg_y;
    
  reg_x.x = fminf(fmaxf(reg_x.x, MIN_EXP_F32), MAX_EXP_F32);
  reg_x.y = fminf(fmaxf(reg_x.y, MIN_EXP_F32), MAX_EXP_F32);
  reg_x.z = fminf(fmaxf(reg_x.z, MIN_EXP_F32), MAX_EXP_F32);
  reg_x.w = fminf(fmaxf(reg_x.w, MIN_EXP_F32), MAX_EXP_F32);

  reg_y.x = 1.0f / (1.0f + expf(-reg_x.x));
  reg_y.y = 1.0f / (1.0f + expf(-reg_x.y));
  reg_y.z = 1.0f / (1.0f + expf(-reg_x.z));
  reg_y.w = 1.0f / (1.0f + expf(-reg_x.w));

  if ((idx + 0) < N) { FLOAT4(y[idx]) = reg_y; }
}

// -------------------------------------- FP16 -------------------------------------- 
__global__ void sigmoid_f16_kernel(half* x, half* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const half f = __float2half(1.0f);
  if (idx < N) {
    half v = x[idx];
    v = __hmin(__hmax(v, MIN_EXP_F16), MAX_EXP_F16);
    y[idx] = f / (f + hexp(-v));
  }
}

__global__ void sigmoid_f16x2_kernel(half* x, half* y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
  const half f = __float2half(1.0f);
  half2 reg_x = HALF2(x[idx]);
  half2 reg_y;
  reg_x.x = __hmin(__hmax(reg_x.x, MIN_EXP_F16), MAX_EXP_F16);
  reg_x.y = __hmin(__hmax(reg_x.y, MIN_EXP_F16), MAX_EXP_F16);

  reg_y.x = f / (f + hexp(-reg_x.x));
  reg_y.y = f / (f + hexp(-reg_x.y));

  if ((idx + 0) < N) { HALF2(y[idx]) = reg_y; }
}

// unpack f16x8
__global__ void sigmoid_f16x8_kernel(half* x, half* y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
  const half f = __float2half(1.0f);

  half2 reg_x_0 = HALF2(x[idx + 0]);
  half2 reg_x_1 = HALF2(x[idx + 2]);
  half2 reg_x_2 = HALF2(x[idx + 4]);
  half2 reg_x_3 = HALF2(x[idx + 6]);

  reg_x_0.x = __hmin(__hmax(reg_x_0.x, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_0.y = __hmin(__hmax(reg_x_0.y, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_1.x = __hmin(__hmax(reg_x_1.x, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_1.y = __hmin(__hmax(reg_x_1.y, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_2.x = __hmin(__hmax(reg_x_2.x, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_2.y = __hmin(__hmax(reg_x_2.y, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_3.x = __hmin(__hmax(reg_x_3.x, MIN_EXP_F16), MAX_EXP_F16);
  reg_x_3.y = __hmin(__hmax(reg_x_3.y, MIN_EXP_F16), MAX_EXP_F16);

  half2 reg_y_0, reg_y_1, reg_y_2, reg_y_3;
  
  reg_y_0.x = f / (f + hexp(-reg_x_0.x));
  reg_y_0.y = f / (f + hexp(-reg_x_0.y));
  reg_y_1.x = f / (f + hexp(-reg_x_1.x));
  reg_y_1.y = f / (f + hexp(-reg_x_1.y));
  reg_y_2.x = f / (f + hexp(-reg_x_2.x));
  reg_y_2.y = f / (f + hexp(-reg_x_2.y));
  reg_y_3.x = f / (f + hexp(-reg_x_3.x));
  reg_y_3.y = f / (f + hexp(-reg_x_3.y));

  if ((idx + 0) < N) { HALF2(y[idx + 0]) = reg_y_0; }
  if ((idx + 2) < N) { HALF2(y[idx + 2]) = reg_y_1; }
  if ((idx + 4) < N) { HALF2(y[idx + 4]) = reg_y_2; }
  if ((idx + 6) < N) { HALF2(y[idx + 6]) = reg_y_3; }
}

// pack f16x8
__global__ void sigmoid_f16x8_pack_kernel(half* x, half* y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
  const half f = __float2half(1.0f);
  // temporary register(memory), .local space in ptx, addressable
  half pack_x[8], pack_y[8]; // 8x16 bits=128 bits.
  // reinterpret as float4 and load 128 bits in 1 memory issue.
  LDST128BITS(pack_x[0]) = LDST128BITS(x[idx]); // load 128 bits
  
  #pragma unroll
  for (int i = 0; i < 8; ++i) {
    half v = __hmin(__hmax(pack_x[i], MIN_EXP_F16), MAX_EXP_F16);
    pack_y[i] = f / (f + hexp(-v));
  }
  // reinterpret as float4 and store 128 bits in 1 memory issue.
  if ((idx + 7) < N) { LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]); }
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define TORCH_BINDING_SIGMOID(packed_type, th_type, element_type, n_elements)    \
void sigmoid_##packed_type(torch::Tensor x, torch::Tensor y) {                   \
  CHECK_TORCH_TENSOR_DTYPE(x, (th_type))                                         \
  CHECK_TORCH_TENSOR_DTYPE(y, (th_type))                                         \
  const int ndim = x.dim();                                                      \
  if (ndim != 2) {                                                               \
    int N = 1;                                                                   \
    for (int i = 0; i < ndim; ++i) { N *= x.size(i); }                           \
    dim3 block(256 / (n_elements));                                              \
    dim3 grid((N + 256 - 1) / 256);                                              \
    sigmoid_##packed_type##_kernel<<<grid, block>>>(                             \
      reinterpret_cast<element_type*>(x.data_ptr()),                             \
      reinterpret_cast<element_type*>(y.data_ptr()), N);                         \
  } else {                                                                       \
    const int S = x.size(0);                                                     \
    const int K = x.size(1);                                                     \
    const int N = S * K;                                                         \
    if ((K/(n_elements)) <= 1024) {                                              \
      dim3 block(K/(n_elements));                                                \
      dim3 grid(S);                                                              \
      sigmoid_##packed_type##_kernel<<<grid, block>>>(                           \
        reinterpret_cast<element_type*>(x.data_ptr()),                           \
        reinterpret_cast<element_type*>(y.data_ptr()), N);                       \
    } else {                                                                     \
      int N = 1;                                                                 \
      for (int i = 0; i < ndim; ++i) { N *= x.size(i); }                         \
      dim3 block(256 / (n_elements));                                            \
      dim3 grid((N + 256 - 1) / 256);                                            \
      sigmoid_##packed_type##_kernel<<<grid, block>>>(                           \
        reinterpret_cast<element_type*>(x.data_ptr()),                           \
        reinterpret_cast<element_type*>(y.data_ptr()), N);                       \
    }                                                                            \
  }                                                                              \
}


TORCH_BINDING_SIGMOID(f32,        torch::kFloat32,    float,    1)
TORCH_BINDING_SIGMOID(f32x4,      torch::kFloat32,    float,    4)
TORCH_BINDING_SIGMOID(f16,        torch::kHalf,       half,     1)
TORCH_BINDING_SIGMOID(f16x2,      torch::kHalf,       half,     2)
TORCH_BINDING_SIGMOID(f16x8,      torch::kHalf,       half,     8)
TORCH_BINDING_SIGMOID(f16x8_pack, torch::kHalf,       half,     8)

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(sigmoid_f32)
  TORCH_BINDING_COMMON_EXTENSION(sigmoid_f32x4)
  TORCH_BINDING_COMMON_EXTENSION(sigmoid_f16)
  TORCH_BINDING_COMMON_EXTENSION(sigmoid_f16x2)
  TORCH_BINDING_COMMON_EXTENSION(sigmoid_f16x8)
  TORCH_BINDING_COMMON_EXTENSION(sigmoid_f16x8_pack)
}
