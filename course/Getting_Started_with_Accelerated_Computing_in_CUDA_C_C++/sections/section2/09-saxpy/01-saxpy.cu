#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 200,000 ns.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void saxpy(int * a, int * b, int * c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;

    //if ( tid < N )
    //    c[tid] = 2 * a[tid] + b[tid];
    
    for (int i = tid; i < N; i += gridStride)
      c[i] = 2 * a[i] + b[i];
}

int main()
{
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    printf("Device ID: %d\tNumber of SMs: %d\n", deviceId, numberOfSMs);

    int *a, *b, *c;

    int size = N * sizeof (int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    hipMemPrefetchAsync(a, size, hipCpuDeviceId);
    hipMemPrefetchAsync(b, size, hipCpuDeviceId);
    hipMemPrefetchAsync(c, size, hipCpuDeviceId);

    // Initialize memory
    for( int i = 0; i < N; ++i )
    {
        a[i] = 2;
        b[i] = 1;
        c[i] = 0;
    }

    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);

    int threads_per_block = 128;
    // int number_of_blocks = (N / threads_per_block) + 1;
    int number_of_blocks = 32*numberOfSMs;

    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );
    hipDeviceSynchronize();

    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}
